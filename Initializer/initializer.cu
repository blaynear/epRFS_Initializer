#include "hip/hip_runtime.h"
#include "initializer.cuh"

using namespace std;
const int baseThreadCount = 64;

__host__ __device__ unsigned int getThreadCount(unsigned int offset){
	return min(((offset / 512) + 1)*baseThreadCount, 512);
}

__host__ __device__ unsigned int getBlockCount(unsigned int offset, unsigned int numberOfThreads){
	return (offset / numberOfThreads) + 1;
}


/***************************************************************************/
/*   											      Cuda Kernels   														 */
/***************************************************************************/
__global__ void setPuzzle(unsigned int *size, Puzzle *aPuzzle){
  aPuzzle[0] = Puzzle(size[0]);
}

__global__ void numeralChanger(unsigned int *size, Puzzle *aPuzzle, unsigned int ***searchSpace){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int row = tid % size[0];
  int col = (tid / size[0]) % size[0];

  for(int i = 0; i < size[0]; i++){
    aPuzzle[0].allocatePuzzle(row, col, i, searchSpace[row][col][i]);
  }
}

__global__ void exterminateRegions(unsigned int *size, unsigned int ***searchSpace, unsigned int aClue){
  double sqrtSize = size[0];
  sqrtSize = sqrt(sqrtSize);

  int tid = threadIdx.x + blockIdx.x * blockDim.x,
    row = aClue % size[0],
	  col = (aClue / size[0]) % size[0],
	  num = aClue / (size[0]*size[0]),
	  aClue_region_row = row / sqrtSize, // Region row 0,1, or 2
	  aClue_region_col = col / sqrtSize;

  if(tid < size[0]){
    //printf("clue: %d tid: %d row: %d col: %d num: %d\n", aClue, tid, row, col, num);
    searchSpace[tid][col][num] = 1;
    searchSpace[row][tid][num] = 1;
    searchSpace[row][col][tid] = 1;
    searchSpace[aClue_region_row * (int)sqrtSize + tid%(int)sqrtSize][aClue_region_col * (int)sqrtSize + tid/(int)sqrtSize][num] = 1;
  }

}


__global__ void exterminate(unsigned int *size, unsigned int *listOfClues, unsigned int ***searchSpace, unsigned int *numClues){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid < numClues[0]){
		hipStream_t stream;
		hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    int threadCount = getThreadCount(size[0]);
    int blockCount = getBlockCount(size[0], threadCount);
    exterminateRegions<<<blockCount, threadCount, 0, stream>>>(size, searchSpace, listOfClues[tid]);
    hipDeviceSynchronize();
		hipStreamDestroy(stream);
  }
}

__global__ void printPuzzleFile(Puzzle *aPuzzle, unsigned int size){

  for (int row = 0; row < size; row++){
		for (int col = 0; col < size; col++){
      if (col == size - 1){
        printf("%u\t\n", aPuzzle[0].getCellNumeral(row, col));
      }else{
        printf("%u, \t", aPuzzle[0].getCellNumeral(row, col));
      }
		}
	}
}

/***************************************************************************/
/*   														Constructors   														 */
/***************************************************************************/

__host__ Initializer::Initializer(unsigned int size){
  this->d_puzzle = allocateDevice<Puzzle>(1);
  this->size = size;
  this->threads = getThreadCount(size * size);
  this->blocks = getBlockCount(size * size, this->threads);
  cudaCheckError();
}

__host__ void Initializer::run(fstream &file){
  int row=0, col=0, num=0, clues=0;
  file >> clues;

  unsigned int *d_size = allocateDevice<unsigned int>(1);
  unsigned int *d_numClues = allocateDevice<unsigned int>(1);
  unsigned int *d_listClues = allocateDevice<unsigned int>(clues);
  
  hipMemcpy(d_size, &this->size, sizeof(unsigned int), hipMemcpyHostToDevice);
  cudaCheckError();
  hipMemcpy(d_numClues, &clues, sizeof(unsigned int), hipMemcpyHostToDevice);
  cudaCheckError();

  unsigned int *listOfClues = allocateHost<unsigned int>(clues);

  int index = 0;
  while (file >> row >> col >> num){
    listOfClues[index] = get3DIndex(row - 1, col - 1, num - 1,  this->size);
    index++;
  }

  hipMemcpy(d_listClues, listOfClues, clues*sizeof(unsigned int), hipMemcpyHostToDevice);
  cudaCheckError();

  unsigned int ***h_searchSpace = allocateHost<unsigned int**>( this->size);
  
  for(int i = 0; i < this->size; i++){
    h_searchSpace[i] = allocateHost<unsigned int*>( this->size);
  }

  for(int i = 0; i < this->size; i++){
    for(int j = 0; j < this->size; j++){
      h_searchSpace[i][j] = allocateHost<unsigned int>( this->size);
    }
  }

  for(int i = 0; i < this->size; i++){
    for(int j = 0; j < this->size; j++){
      for(int k = 0; k < this->size; k++){
        h_searchSpace[i][j][k] = 0;
      }
    }
  }

  unsigned int ***d_searchSpace = allocateDevice<unsigned int**>(this->size);
  hipMemcpy(d_searchSpace, h_searchSpace, this->size*sizeof(unsigned int**), hipMemcpyHostToDevice);
  cudaCheckError();
  
  int threadCount = getThreadCount(clues);
  int blockCount = getBlockCount(clues, threadCount);//launch clueNumber blocks and threads

  exterminate<<<blockCount,threadCount>>>(d_size, d_listClues, d_searchSpace, d_numClues);
  cudaCheckError();

  hipMemcpy(h_searchSpace, d_searchSpace, this->size*sizeof(unsigned int**), hipMemcpyDeviceToHost);
  cudaCheckError();
  
  /*
  int numSpecies = 0;
  for(int i = 0; i < this->size; i++){
    for(int j = 0; j < this->size; j++){
      for(int k = 0; k < this->size; k++ ){
        if(h_searchSpace[i][j][k] == 1){numSpecies++;}
      }
    }
  }
  numSpecies = ( this->size * this->size * this->size)-numSpecies;
  */
  
  setPuzzle<<<1,1>>>(d_size, d_puzzle);
  numeralChanger<<<this->blocks,this->threads>>>(d_size, d_puzzle, d_searchSpace);
  cudaCheckError();

}

__host__ void Initializer::printPuzzle(){
  printf("\nSudoku Puzzle Printout\n");
  this->h_puzzle = allocateHost<Puzzle>(1);

  hipMemcpy(h_puzzle, this->d_puzzle, sizeof(Puzzle), hipMemcpyDeviceToHost);
  cudaCheckError();

  printPuzzleFile<<<1,1>>>(h_puzzle, this->size);
}



__host__ __device__ unsigned int Initializer::get3DIndex(int x, int y, int z, int dim) {
	return x + dim*(y + dim*z);
}
/***************************************************************************/
/*														Getter Functions														 */
/***************************************************************************/

__host__ __device__ unsigned int Initializer::getSize(){
	return this->d_puzzle[0].getSize();
  }

/***************************************************************************/
/*														Setter Functions														 */
/***************************************************************************/

/***************************************************************************/
/*														CUDA Helper Masks														 */
/***************************************************************************/

template<class t>
t *Initializer::allocateDevice(unsigned int size){
	t *aValue;
	hipMalloc((void **)&aValue, size * sizeof(t));
	cudaCheckError();
	return aValue;
}

template<class t>
t *Initializer::allocateHost(unsigned int size){
	t *aValue;
	hipHostMalloc((void **)&aValue, size * sizeof(t));
	cudaCheckError();
	return aValue;
}



/*template <unsigned int blockSize, int function>
__global__ void getConflictData(Puzzle dPuzzle, int numSpecies){
	extern __shared__ unsigned int sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + tid;

	unsigned int offset = numSpecies/2; //get numSpecies

	if(numSpecies % 2 == 1){  
		offset++;
	}

	while(i < offset){
		sdata[tid] += aPuzzle.population[i].conflictSize + aPuzzle.population[i + offset].conflictSize;
		i += blockSize;
	}

	__syncthreads();
	blockReduce<blockSize, function, 512, 256>(sdata, tid);
	blockReduce<blockSize, function, 256, 128>(sdata, tid);
	blockReduce<blockSize, function, 128, 64>(sdata, tid);
	if (tid < 32){
		warpReduce<blockSize, function>(sdata, tid);
	}
	if(tid == 0){
				atomicAdd(&params->totalConflicts, sdata[0]);
				params->avgConflicts = params->totalConflicts / numSpecies + 1;
	}
}

template <unsigned int blockSize, int function, unsigned int blockBoundary, unsigned int tidMax>
__device__ void blockReduce(volatile unsigned int *sdata, unsigned int tid){
	if(blockSize >= blockBoundary){
		if (tid < tidMax){
			conflictOperation<function, tidMax>(sdata, tid);
		}
		__syncthreads();
	}
}

template <unsigned int blockSize, int function>
__device__ void warpReduce(volatile unsigned int *sdata, unsigned int tid){
	if(blockSize >= 64){
		conflictOperation<function, 32>(sdata, tid);
	}
	if (blockSize >= 32){
		conflictOperation<function, 16>(sdata, tid);
	}
	if (blockSize >= 16){
		conflictOperation<function, 8>(sdata, tid);
	}
	if(blockSize >= 8){
		conflictOperation<function, 4>(sdata, tid);
	}
	if (blockSize >= 4){
		conflictOperation<function, 2>(sdata, tid);
	}
	if (blockSize >= 2){
		conflictOperation<function, 1>(sdata, tid);
	}
}

template <int function, unsigned int offset>
__device__ void conflictOperation(volatile unsigned int *sdata, unsigned int tid){
	sdata[tid] += sdata[tid + offset];
}*/
