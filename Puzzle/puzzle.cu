#include "hip/hip_runtime.h"
#include "puzzle.cuh"

const int baseThreadCount = 64;

/***************************************************************************/
/*   											      Cuda Kernels   														 */
/***************************************************************************/

/***************************************************************************/
/*   														Constructors   														 */
/***************************************************************************/
__device__ void Puzzle::allocatePuzzle(int x, int y, unsigned int index, unsigned int set){
	this->grid[x][y].setNumeral(index, set);
}

__host__ __device__ Puzzle::Puzzle(unsigned int size){
	this->dim = size;
	this->grid = allocateDevice<Cell *>(size);
	for(int i = 0; i < size; i++){
		this->grid[i] = allocateDevice<Cell>(size);
	}
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			this->grid[i][j] = Cell(size);
		}
	}
}

/***************************************************************************/
/*						Getter Functions   								   */
/***************************************************************************/
__host__ __device__ unsigned int Puzzle::getSize(){
  return this->dim;
}

__host__ __device__ unsigned int Puzzle::getCellSize(unsigned int x, unsigned int y){
	return this->grid[x][y].getSize();
}

__host__ __device__ unsigned int Puzzle::getCellNumeral(unsigned int x, unsigned int y){
	unsigned int numProp;
	numProp = this->grid[x][y].getNumeral();

	return numProp;
}

/***************************************************************************/
/*														CUDA Helper Masks														 */
/***************************************************************************/
template<class t>
__host__ __device__ t *Puzzle::allocateHost(unsigned int size){
	t *aValue;
	hipHostMalloc((void **)&aValue, size * sizeof(t));
	cudaCheckError();
	return aValue;
}

template<class t>
__host__ __device__ t *Puzzle::allocateDevice(unsigned int size){
	t *aValue;
	hipMalloc((void **)&aValue, size * sizeof(t));
	cudaCheckError();
	return aValue;
}

__host__ __device__ unsigned int Puzzle::getThreadCount(){
	this->threads = min((((this->dim*this->dim) / 512) + 1)*baseThreadCount, 512);
	return this->threads;
}

__host__ __device__ unsigned int Puzzle::getBlockCount(){
	this->blocks = ((this->dim*this->dim) / this->threads) + 1;
	return this->blocks;
}

__host__ __device__ unsigned int Puzzle::getRow(int index, int dim){
	return index % dim;
}

__host__ __device__ unsigned int Puzzle::getCol(int index, int dim){
	return (index / dim) % dim;
}

__host__ __device__ unsigned int Puzzle::getNum(int index, int dim){
	return index / (dim*dim);
}

__host__ __device__ unsigned int Puzzle::getSquare(int index, int dim){
	return get2DIndex(getRow(index, dim) / sqrtf((double)dim), getCol(index, dim) / sqrtf((double)dim), sqrtf((double)dim));
}

__host__ __device__ unsigned int Puzzle::get3DIndex(int x, int y, int z, int dim) {
	return x + dim*(y + dim*z);
}

__host__ __device__ unsigned int Puzzle::get2DIndex(int x, int y, int dim) {
	return x + dim*y;
}

__host__ __device__ void Puzzle::free_puzzle(){
	//grid.free_cell();
	hipFree(grid);
}